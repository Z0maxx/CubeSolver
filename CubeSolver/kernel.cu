﻿#include "cube_solver.cuh"
#include "variables.cuh"
#include "sequence_processor.cuh"
#include "notation_translator.cuh"
#include "test_rotations.cuh"

int main()
{
    hipSetDeviceFlags(hipDeviceScheduleSpin);
    long long start = timeInMilliseconds();
    solve(testCubeColors);
    findSequence();
    Notation sequence[263]{};
    hipMemcpyFromSymbol(sequence, HIP_SYMBOL(dev_sequence), sizeof(sequence));
    long long end = timeInMilliseconds();
    printf("%d\n", end - start);
    for (int i = 0; i < 263; i++)
    {
        if (sequence[i] != None)
        {
            const TranslatedNotation move = translateNotation(sequence[i]);
            if (move.layer)
            {
                turnTestLayer(move.cubeLayer, move.direction, move.twice);
            }
            else
            {
                turnTestCube(move.direction, move.twice);
            }
            printf("%d ", sequence[i]);
        }
    }
    Color testColors[3][9][6]{};
    for (int i = 0; i < 3; i++)
    {
        for (int j = 0; j < 9; j++)
        {
            for (int k = 0; k < 6; k++)
            {
                testColors[i][j][k] = (Color)testCubeColors[i][j][k];
            }
        }
    }
    printf("\n");
    printCube(testColors);
    scanf("%s");
}
